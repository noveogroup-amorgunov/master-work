#include "hip/hip_runtime.h"
/********************************************************************
*  sample.cu
*  This is a example of the CUDA program.
*********************************************************************/

#include <stdio.h>
#include <stdlib.h>
//#include <cutil_inline.h>
#include <iostream>
#include <sstream>
#include <vector>
//#include "conio.h"
#include <fstream> // File-stream
#include <string>
#include <map> 
#include <omp.h>
#include <time.h>
#include <list>
#include <hipblas.h>



#include "mt.h"
//#include "SAnTimer.h"
//#include "Debug.h"

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}

#endif
/************************************************************************/
/* CUDA                                                             */
/************************************************************************/

__global__ static void HelloCUDA(char* result, int num)
{
	int i = 0;
	char p_HelloCUDA[] = "Hello CUDA!";
	for(i = 0; i < num; i++) {
		result[i] = p_HelloCUDA[i];
	}
}

__global__ void StatCount(float* real, float* rand, int* mas1, int* mas2, int* mas3, int* mas4, int* mas5, int temp_0gen)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while ( tid < temp_0gen)//
	{
		if (real[tid] <  rand[tid]) mas1[tid]++;
		if (real[tid] <= rand[tid]) mas2[tid]++;
		if (real[tid] >  rand[tid]) mas3[tid]++;
		if (real[tid] >= rand[tid]) mas4[tid]++;
		if (real[tid] == rand[tid]) mas5[tid]++;
		tid += blockDim.x * gridDim.x;
	}
	__syncthreads();
}

//*******************************************************************************

// Функция для разделения строки s на подстроки по символу delim. Результат в векторе elems
std::vector<std::string> &split (const std::string &s, char delim, std::vector<std::string> &elems)
{
	std::stringstream ss(s);
	std::string item;
	//std::string tmp = " ";
	while(std::getline(ss, item, delim))
	{
		if(item !="")// && item !=tmp.c_str()) //.c_str()
		{
			for (std::string::iterator it = item.begin() ; it<item.end(); ++it )
			{
				if (*it == ' ') item.erase(it);
			}
			elems.push_back(item);
		}
	 }
	//split (elems.at(0), ' ', elems);
	return elems;
}

// Аналогично предыдущей функции
void tokenize(const std::string& str, std::vector<std::string>& tokens,\
              const std::string& delimiters = " ", const bool trimEmpty = true)
{
	std::string::size_type pos, lastPos = 0;
	while(true)
	{
		pos = str.find_first_of(delimiters, lastPos);
		if(pos == std::string::npos)
		{
			pos = str.length();
			if(pos != lastPos || !trimEmpty)
			{
				tokens.push_back(std::string(str.data()+lastPos,(std::string::size_type)pos-lastPos ));
			}
			break;
		}
		else
		{
			if(pos != lastPos || !trimEmpty)
			{
				tokens.push_back(std::string(str.data()+lastPos,(std::string::size_type)pos-lastPos ));
			}
		}
		lastPos = pos + 1;
	}
};

using namespace std;

/************************************************************************/
/* HelloCUDA                                                            */
/************************************************************************/
int main(int argc, char* argv[])
{

	if(!InitCUDA()) {
		return 0;
	}
	string str;
	string filename;
	string name_delim;//Делиметер ФА
	string stps,smp_sub_stp; // Количество итераций и количество перестановок
	int temp_0gen = 0;
	int temp_1gen = 0;

	//timer::Timer t_all(true); //Создаём таймер всей работы программы и сразу его запускаем
	//timer::Timer t_file(true); //Создаём таймер обработки файла и сразу его запускаем
	//Чтение файла и разбор файла
	//===================================================================
	int count_str = 0; //количество строк в файле
	ifstream file,file1,conf;//

	conf.open( "conf.txt" ); // открываем файл для чтения настроек
	if(file == NULL)
	{
		printf("file conf.txt not found");
		getchar();
		return EXIT_SUCCESS;
	}
	else
	{
			getline(conf,filename);
			getline(conf,name_delim);
			getline(conf,stps);
			getline(conf,smp_sub_stp);
	}
	conf.close();

	file.open("input.txt");// открываем файл первый раз для подсчета кол-ва строк
	if(file == NULL) 
	{
    printf("file not found");
		printf("%s", filename.c_str());
		getchar();
		return EXIT_SUCCESS;
	}
	else
	{
		while( getline(file,str) )//пока есть строки
		{
			count_str++; //счет строк
		}
	}
	string *arr = new string[count_str];// массив для строк
	int *GEN_ID_arr = new int[count_str-1]; // массив идентификаторов строк
	float *ValueGen_arr = new float[count_str-1]; // массив значений генов //new
	
	file.close();//закрыли файл

	cout << " Start parsing file! " << endl;
	
	file1.open( "input.txt" );// открыли снова для разбора строк по столбцам
	
	int id = 0;
	while( getline(file1,str, '\n') )//пока есть строки читаем строки в str
		{
			arr[id] = str; //помещаем строки в массив
			id++;
		}

	file1.close();

	std::vector <std::vector<std::string> > all_term_gen;//Двумерный вектор с генами строк по строкам
	all_term_gen.resize(count_str-1); // Первое измерение по кол-ву рабочих строк
	
	map <string, std::list<std::string> > FA_in_gen;// вхождения ФА с ненулевыми значениями в гены пара(ФА, вектор и ид генов)
	map <string, std::list<std::string> > ::iterator IT_FA_in_gen;// итератор для мапа 
	map <string, std::list<std::string> > FA_in_gen_all;// вхождения ФА в гены пара(ФА, вектор и ид генов)
	map <string, std::list<std::string> >::iterator IT_FA_in_gen_all;// итератор для мапа 
 

	map<string, float> real_val_sum_for_term;// накопление значений на ген пара(ген, значение гена)
	map<string, int> sum_of_gens_for_term;// количество генов пара(ген, количество)
	map<string, float>::iterator IT_real_val_sum_for_term;// итератор для мапа по значением генов
	map<string, int>::iterator IT_sum_of_gens_for_term;// итератор для мапа по количеству генов

	map<string, int> gen_in_term;// набор генов со значением больше ноля пара(ген, индекс строки)
	map<int, float> value_for_gen;// значение гена пара(индекс строки, значение гена)
	map<string, int>::iterator IT_gen_in_term;// итератор для мапа 
	map<int, float>::iterator IT_value_for_gen;// итератор для мапа

	map<string, int> all_gen_in_term;//набор генов пара(ген, индекс строки)
	map<string, int>::iterator IT_all_gen_in_term;// итератор для мапа

	map<int, int> size_map;// мап с парой (индекс строки, кол-во генов в ней)
	map<int, int>::iterator IT_size_map;// итератор для мапа

	for(int i = 1;i<count_str;i++)// i=1 Пропускаем первую строчку с заголовком и по оставшимся идем
	{
		std::vector<std::string> elems;
		split (arr[i], '\t', elems); // дробим строчку на составляющие по пробелу (пробелы вырезаются)
		std::vector<std::string> tmp_gen_terms;//временное хранилище генов
		tokenize(elems.at(1), tmp_gen_terms,name_delim);
		size_map[atoi(elems.at(0).c_str())] = tmp_gen_terms.size(); // мап с количеством генов по строкам
		all_term_gen[i-1].resize(tmp_gen_terms.size());// второе измерение по кол-ву генов

		for(unsigned int k=0; k<tmp_gen_terms.size(); k++) // 
		{
			all_term_gen[i-1][k] = tmp_gen_terms.at(k);// складываем гены в вектор соответствующей строки
			real_val_sum_for_term[tmp_gen_terms.at(k)] += (float)(atof(elems.at(elems.size()-1).c_str()));
			sum_of_gens_for_term[tmp_gen_terms.at(k)] += 1;
			
			if(all_gen_in_term.count(tmp_gen_terms.at(k)) != 1)
			{
				all_gen_in_term[tmp_gen_terms.at(k)]=(int)(atoi(elems.at(0).c_str()));//!!!
				temp_0gen++;
			}

			if (FA_in_gen_all.count(tmp_gen_terms.at(k))!= 1)
			{
				//создаем список
				list<string> lst;
				// добавляем в него идентификатор строки;
				lst.push_back(elems.at(0));
				FA_in_gen_all[tmp_gen_terms.at(k)] = lst;
			}
			else
			{
				(*FA_in_gen_all.find(tmp_gen_terms.at(k))).second.push_back(elems.at(0));
			}

			if((float)(atof(elems.at(elems.size()-1).c_str()))>0)
			{
				//FA_in_gen формируем с списком строк с ФА с ненулевым значением
				//---------------------
				if (FA_in_gen.count(tmp_gen_terms.at(k))!= 1)
				{
					//создаем список
					list<string> lst;
					// добавляем в него идентификатор строки;
					lst.push_back(elems.at(0));
					FA_in_gen[tmp_gen_terms.at(k)] = lst;
				}
				else
				{
					(*FA_in_gen.find(tmp_gen_terms.at(k))).second.push_back(elems.at(0));
				}
				//--------------------
				if(gen_in_term.count(tmp_gen_terms.at(k)) != 1)
				{
					gen_in_term[tmp_gen_terms.at(k)]=(int)(atoi(elems.at(0).c_str()));//!!!
					temp_1gen++;
				}
			}
		}
		value_for_gen[(int)(atoi(elems.at(0).c_str()))] = (float)(atof(elems.at(elems.size()-1).c_str()));
		GEN_ID_arr[i-1] = (int)(atoi(elems.at(0).c_str()));
		ValueGen_arr[i-1] = (float)(atof(elems.at(elems.size()-1).c_str()));// new
	}

	float *SqData = new float[temp_0gen*(count_str-2)];//двумерный массив, вытянутый в одномерный, хранящий 0 или 1 вхождения ФА [countFA * countStr]
	for (int i = 0; i < temp_0gen*(count_str-2); i++)
	{
		SqData[i] = 0.0;
	}
	//Замечание 
	//j*str+i - транспонированный вид ФА*Стр
	//temp_0gen*i+j - Нормальный вид Стр*ФА
	for (int i = 0; i < count_str-2; i++)
	{
		//test << "----i " << i << endl;
		int j = 0;
		//int str = count_str-2;
		for (IT_all_gen_in_term = all_gen_in_term.begin();\
			IT_all_gen_in_term != all_gen_in_term.end();\
			IT_all_gen_in_term++)
		{
			//test << "--j " << j << endl;
			for (int k = 0; k < all_term_gen[i].size(); k++)
			{
				//test << "k " << k;
				if ((*IT_all_gen_in_term).first == all_term_gen[i][k])
				{
					SqData[temp_0gen*i+j] = 1.0;// temp_0gen*i+j j*str+i
				}
			}
			j++;
		}
	}

	//t_file.pause();
	cout << " Parsing file complete! " << endl;
	//cout << "Time to parse the file = " << t_file << " seconds." << endl;
	
	int steps = atoi(stps.c_str());
	int subsample_size = atoi(smp_sub_stp.c_str());
	
	if (subsample_size > count_str-1) 
	{
		cout << " Incorrect fourth parameters!!! Now he is " << count_str-1 << endl;
		subsample_size = count_str-1;
	}

	// Initialize a Mersenne Twister
	MersenneTwister mt;
	
	cout << " Start shuffle! " << endl;
	
	//timer::Timer t_shuffle(true); //Создаём таймер и сразу его запускаем

	//mt.init_genrand((unsigned long)t_shuffle.getTickCount());
	//Весь цикл расчета
	//timer::Timer t_3(false); //
	//timer::Timer t_2(false); //
	//timer::Timer t_1(false); //



	//Создаем массивы для вычисления рандомной суммы на ГПУ
	float *d_AA, *d_x, *d_Ax, *d_AxReal;//, *d_test;
	int size_str = (count_str-2);
	int size_Full = temp_0gen*size_str;
	
	/*
	cutilSafeCall( hipMalloc((void**) &d_AxReal, temp_0gen*sizeof(float)) ); //Реальные суммы
	cutilSafeCall( hipMalloc((void**) &d_x, size_str*sizeof(float)) ); // Вектор значений
	cutilSafeCall( hipMalloc((void**) &d_Ax, temp_0gen*sizeof(float)) );// Случайные суммы
	cutilSafeCall( hipMalloc((void**) &d_AA, size_Full*sizeof(float)) );// Матрица вхождений ФА
	cutilSafeCall( hipMemcpy(d_AA, SqData, size_Full*sizeof(float), hipMemcpyHostToDevice) );
	*/
	
	hipMalloc((void**) &d_AxReal, temp_0gen*sizeof(float)); //Реальные суммы
	hipMalloc((void**) &d_x, size_str*sizeof(float)); // Вектор значений
	hipMalloc((void**) &d_Ax, temp_0gen*sizeof(float));// Случайные суммы
	hipMalloc((void**) &d_AA, size_Full*sizeof(float));// Матрица вхождений ФА
	hipMemcpy(d_AA, SqData, size_Full*sizeof(float), hipMemcpyHostToDevice);
	
	//Создаем массивы для насчета количеств вхождений по рандомной сумме
	int *sum_of_cases_with_more_values_sum_arr,\
		*sum_of_cases_with_more_or_equal_values_sum_arr,\
		*sum_of_cases_with_less_values_sum_arr,\
		*sum_of_cases_with_less_or_equal_values_sum_arr,\
		*sum_of_cases_with_equal_values_sum_arr;
		
	float *AxReal, *Ax;
	
	sum_of_cases_with_more_values_sum_arr = new int[temp_0gen];
	sum_of_cases_with_more_or_equal_values_sum_arr = new int[temp_0gen];
	sum_of_cases_with_less_values_sum_arr = new int[temp_0gen];
	sum_of_cases_with_less_or_equal_values_sum_arr = new int[temp_0gen];
	sum_of_cases_with_equal_values_sum_arr = new int[temp_0gen];
	
	AxReal = new float[temp_0gen];
	Ax = new float[temp_0gen];
	
	for(int i = 0; i < temp_0gen; i++)
	{
		sum_of_cases_with_more_values_sum_arr[i] = 0;
		sum_of_cases_with_more_or_equal_values_sum_arr[i] = 0;
		sum_of_cases_with_less_values_sum_arr[i] = 0;
		sum_of_cases_with_less_or_equal_values_sum_arr[i] = 0;
		sum_of_cases_with_equal_values_sum_arr[i] = 0;
		AxReal[i] = 0.0f;
		Ax[i] = 0.0f;
	}

	sum_of_cases_with_more_values_sum_arr = (int*) calloc (temp_0gen,sizeof(int));
	sum_of_cases_with_more_or_equal_values_sum_arr = (int*) calloc (temp_0gen,sizeof(int));
	sum_of_cases_with_less_values_sum_arr = (int*) calloc (temp_0gen,sizeof(int));
	sum_of_cases_with_less_or_equal_values_sum_arr = (int*) calloc (temp_0gen,sizeof(int));
	sum_of_cases_with_equal_values_sum_arr = (int*) calloc (temp_0gen,sizeof(int));

	int *d_sum_of_cases_with_more_values_sum,\
		*d_sum_of_cases_with_more_or_equal_values_sum,\
		*d_sum_of_cases_with_less_values_sum,\
		*d_sum_of_cases_with_less_or_equal_values_sum,\
		*d_sum_of_cases_with_equal_values_sum;
	
	/*
	cutilSafeCall( hipMalloc((void**)&d_sum_of_cases_with_more_values_sum, temp_0gen*sizeof(int)) );
	cutilSafeCall( hipMalloc((void**)&d_sum_of_cases_with_more_or_equal_values_sum, temp_0gen*sizeof(int)) );
	cutilSafeCall( hipMalloc((void**)&d_sum_of_cases_with_less_values_sum, temp_0gen*sizeof(int)) );
	cutilSafeCall( hipMalloc((void**)&d_sum_of_cases_with_less_or_equal_values_sum, temp_0gen*sizeof(int)) );
	cutilSafeCall( hipMalloc((void**)&d_sum_of_cases_with_equal_values_sum, temp_0gen*sizeof(int)) );

	cutilSafeCall( hipMemcpy(d_sum_of_cases_with_more_values_sum,			sum_of_cases_with_more_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(d_sum_of_cases_with_more_or_equal_values_sum, sum_of_cases_with_more_or_equal_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(d_sum_of_cases_with_less_values_sum,			sum_of_cases_with_less_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice) ); 
	cutilSafeCall( hipMemcpy(d_sum_of_cases_with_less_or_equal_values_sum, sum_of_cases_with_less_or_equal_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(d_sum_of_cases_with_equal_values_sum,			sum_of_cases_with_equal_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(d_AxReal, AxReal, temp_0gen*sizeof(float), hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(d_Ax, Ax, temp_0gen*sizeof(float), hipMemcpyHostToDevice) );
	*/
	
	
	
	

	hipMalloc((void**)&d_sum_of_cases_with_more_values_sum, temp_0gen*sizeof(int));
	hipMalloc((void**)&d_sum_of_cases_with_more_or_equal_values_sum, temp_0gen*sizeof(int));
	hipMalloc((void**)&d_sum_of_cases_with_less_values_sum, temp_0gen*sizeof(int));
	hipMalloc((void**)&d_sum_of_cases_with_less_or_equal_values_sum, temp_0gen*sizeof(int));
	hipMalloc((void**)&d_sum_of_cases_with_equal_values_sum, temp_0gen*sizeof(int));

	hipMemcpy(d_sum_of_cases_with_more_values_sum,			sum_of_cases_with_more_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_of_cases_with_more_or_equal_values_sum, sum_of_cases_with_more_or_equal_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_of_cases_with_less_values_sum,			sum_of_cases_with_less_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice); 
	hipMemcpy(d_sum_of_cases_with_less_or_equal_values_sum, sum_of_cases_with_less_or_equal_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_of_cases_with_equal_values_sum,			sum_of_cases_with_equal_values_sum_arr, temp_0gen*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_AxReal, AxReal, temp_0gen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Ax, Ax, temp_0gen*sizeof(float), hipMemcpyHostToDevice);
	
	int sG,sB;
	if (temp_0gen < 128)
	{
		sG = 1;
		sB = temp_0gen;
	}
	else
	{
		sG = temp_0gen/128 + 1;
		sB = 128;
	}

	// Вычисляем сумму значений по каждой ФА и результат в d_Ax
	// Заводим массив с реальными значениями сумм по каждой ФА				!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	//cutilSafeCall( hipMemcpy(d_x, ValueGen_arr, size_str*sizeof(float), hipMemcpyHostToDevice) );
	hipMemcpy(d_x, ValueGen_arr, size_str*sizeof(float), hipMemcpyHostToDevice);
	const float alpha = 1.0f;
    const float beta = 0.0f;
	hipblasSgemv('n', temp_0gen, size_str, alpha, d_AA, temp_0gen, d_x, 1, beta, d_AxReal, 1);

	for(int stp = 0;stp < steps;stp++)
	{
		//Цикл перестановки
		//t_1.start();
		//mt.init_genrand((unsigned long)t_1.getTickCount());
		for(int sample_stp = 0; sample_stp < subsample_size; sample_stp++)
		{
			int pos = (int)((count_str-2)*mt.genrand_res53());//+0.5);
			int pos2 = (int)((count_str-2)*mt.genrand_res53());//+0.5);
			float tmp = ValueGen_arr[pos2];
			ValueGen_arr[pos2] = ValueGen_arr[pos];
			ValueGen_arr[pos] = tmp;
		}
		//Копируем перемешанный вектор значений на GPU
		//cutilSafeCall( hipMemcpy(d_x, ValueGen_arr, size_str*sizeof(float), hipMemcpyHostToDevice) );
		hipMemcpy(d_x, ValueGen_arr, size_str*sizeof(float), hipMemcpyHostToDevice);
		//Конец Копируем перемешанный вектор значений на GPU

		//t_1.pause();//
		//t_2.start();

		// -----------  Насчет суммы Этап 2  -----------------
		// Вычисляем сумму значений по каждой ФА и результат в d_Ax, перед этим обнулив d_Ax
		hipblasSgemv('N', temp_0gen, size_str, alpha, d_AA, temp_0gen, d_x, 1, beta, d_Ax, 1);
		//-------------------------------------
		//t_2.pause();//
		//t_3.start();
		// -----------  Этап 3. Насчет количеств -----------------
		\

		// завести выше фора 5 массивов соответствующих +
		// завести выше фора 5 массивов соответствующих на ГПУ +
		// аллоцировать на ГПУ 0 +
		// Написать функцию для ГПУ считающую количества
		// после фора скопировать эти массивы на хост+
		StatCount<<<sG, sB>>>(d_AxReal, d_Ax, d_sum_of_cases_with_more_values_sum,\
			d_sum_of_cases_with_more_or_equal_values_sum,\
			d_sum_of_cases_with_less_values_sum,\
			d_sum_of_cases_with_less_or_equal_values_sum,\
			d_sum_of_cases_with_equal_values_sum,\
			temp_0gen) ;
		hipDeviceSynchronize();
		// ---------------------------------
		//t_3.pause();//
		
	}//Конец всего цикла расчета

	/*
	cutilSafeCall( hipMemcpy(sum_of_cases_with_more_values_sum_arr, d_sum_of_cases_with_more_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(sum_of_cases_with_more_or_equal_values_sum_arr, d_sum_of_cases_with_more_or_equal_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(sum_of_cases_with_less_values_sum_arr, d_sum_of_cases_with_less_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(sum_of_cases_with_less_or_equal_values_sum_arr, d_sum_of_cases_with_less_or_equal_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(sum_of_cases_with_equal_values_sum_arr, d_sum_of_cases_with_equal_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost) );
	*/
	
	hipMemcpy(sum_of_cases_with_more_values_sum_arr, d_sum_of_cases_with_more_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(sum_of_cases_with_more_or_equal_values_sum_arr, d_sum_of_cases_with_more_or_equal_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(sum_of_cases_with_less_values_sum_arr, d_sum_of_cases_with_less_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(sum_of_cases_with_less_or_equal_values_sum_arr, d_sum_of_cases_with_less_or_equal_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(sum_of_cases_with_equal_values_sum_arr, d_sum_of_cases_with_equal_values_sum , temp_0gen*sizeof(int), hipMemcpyDeviceToHost);
	
	
	//t_shuffle.pause();
	
	cout << " Shuffle complete!"<<endl;
	//cout << "Time to permutation of rows = " << t_shuffle << " seconds." << endl;
	//cout << " shuffle time " << t_1 << endl;//
	//cout << " Calculate sum " << t_2 << endl;//
	//cout << " Calculate counting " <<t_3 << endl;//

	
	//timer::Timer t_stat(true);

	float *bArr, *bEArr, *sArr, *sEArr, *eArr;
	bArr = new float[temp_0gen];
	bEArr = new float[temp_0gen];
	sArr = new float[temp_0gen];
	sEArr = new float[temp_0gen];
	eArr = new float[temp_0gen];
	for (int i = 0; i < temp_0gen; i++)
	{
		if(sum_of_cases_with_more_values_sum_arr[i] > 0)
		{
			bArr[i] = (float)sum_of_cases_with_more_values_sum_arr[i]/(float)steps;
		}
		else 
		{
			bArr[i] = 1.f/(float)(steps+1);
		}

		if(sum_of_cases_with_more_or_equal_values_sum_arr[i] > 0)
		{
			bEArr[i] = (float)sum_of_cases_with_more_or_equal_values_sum_arr[i]/(float)steps;
		}
		else 
		{
			bEArr[i] = 1.f/(float)(steps+1);
		}

		if(sum_of_cases_with_less_values_sum_arr[i] > 0)
		{
			sArr[i] = (float)sum_of_cases_with_less_values_sum_arr[i]/(float)steps;
		}
		else 
		{
			sArr[i] = 1.f/(float)(steps+1);
		}

		if(sum_of_cases_with_less_or_equal_values_sum_arr[i] > 0)
		{
			sEArr[i] = (float)sum_of_cases_with_less_or_equal_values_sum_arr[i]/(float)steps;
		}
		else 
		{
			sEArr[i] =1.f/(float)(steps+1);
		}

		if(sum_of_cases_with_equal_values_sum_arr[i] > 0)
		{
			eArr[i] = (float)sum_of_cases_with_equal_values_sum_arr[i]/(float)steps;
		}
		else 
		{
			eArr[i] = 1.f/(float)(steps+1); 
		}
	}

	//t_stat.pause();
	//cout << "Time to gather statistics = " << t_stat << " seconds." << endl;

	ofstream out;
	out.open("output.txt");

	//t_all.pause();
	

	if(out == NULL) printf("file not found");
	else
	{
		out << "Count of string in file = "<< count_str - 1 <<endl;
		out << "Number of steps = "<< steps <<endl;
		out << "Size of sub sample = "<< subsample_size <<endl;
		out << "Count of FA = "<< temp_0gen <<endl;
		out << "Count of FA with non-zero value = "<< temp_1gen <<endl;
	//	out << "Full execution time = " << t_all << " seconds." << endl;
	//	out << "Time to parse the file = " << t_file << " seconds." << endl;
	//	out << "Time to permutation of rows = " << t_shuffle << " seconds." << endl;
	//	out << "Time to gather statistics = " << t_stat << " seconds." << endl;
		out << "----------Begin stat info----------"<< endl;
		out << "FA" << '\t'<< "No of All IDs" << '\t' << "No of IDs with NZ" << '\t'<<" s " << '\t' << " e " << '\t' << " b " << '\t' << " sE " << '\t' << " bE " <<endl;
		
		int ci = 0;
		for (IT_all_gen_in_term = all_gen_in_term.begin();IT_all_gen_in_term != all_gen_in_term.end();IT_all_gen_in_term++)
		{
			if(gen_in_term.count((*IT_all_gen_in_term).first) == 1)
			{
				out  << (*IT_all_gen_in_term).first << '\t'; //Имя ФА
				out  << (*FA_in_gen_all.find((*IT_all_gen_in_term).first)).second.size() << '\t';// Число генов с ФА
				out  << (*real_val_sum_for_term.find((*IT_all_gen_in_term).first)).second << '\t'; // Сумма значений генов с ФА
				out  << sArr[ci] << '\t'; 
				out  << eArr[ci] << '\t';
				out  << bArr[ci] << '\t';
				out  << sEArr[ci] << '\t';
				out  << bEArr[ci] << '\t';
				//out  << show(-1) << (*FA_in_gen.find((*IT_all_gen_in_term).first)).second << endl;
				}
			ci++;
		}
		out << "--------End File--------" << endl;
	}
	out.close();

	//-------------------Удаление объектов---------------------
	delete[] GEN_ID_arr;
	delete[] arr;
	hipFree( d_AA );
	hipFree( d_x );
	hipFree( d_Ax );
	hipFree( d_AxReal );
	hipFree( d_sum_of_cases_with_more_values_sum );
	hipFree( d_sum_of_cases_with_more_or_equal_values_sum );
	hipFree( d_sum_of_cases_with_less_values_sum );
	hipFree( d_sum_of_cases_with_less_or_equal_values_sum );
	hipFree( d_sum_of_cases_with_equal_values_sum );
	//hipFree( );
	//----------------Конец Удаления объектов------------------

	//cout << "Full execution time = " << t_all << " seconds." << endl;
	cout << " END! " << endl;
  printf ("success");
	//=========================================================
	//getchar();
  return EXIT_SUCCESS;

	//return 0;
}
